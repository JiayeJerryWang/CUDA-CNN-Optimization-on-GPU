#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
#define MASK_WIDTH 3
#define TILE_WIDTH 8
#define TILE_SIZE TILE_WIDTH + 2
//@@ Define constant memory for device kernel here
__constant__ float Mc[MASK_WIDTH][MASK_WIDTH][MASK_WIDTH];
// Implement a 3D convolution with a 3x3x3 kernel in constant memory and a 3D shared-memory tiling
__global__ void conv3d(float *input, float *output, const int z_size,
                       const int y_size, const int x_size) {
  //@@ Insert kernel code here
  __shared__ float tile[TILE_SIZE][TILE_SIZE][TILE_SIZE];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int bz = blockIdx.z;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int col_o = bx * TILE_WIDTH + tx;
  int row_o = by * TILE_WIDTH + ty;
  int height_o = bz * TILE_WIDTH + tz;
  int col_i = col_o - 1;
  int row_i = row_o - 1;
  int height_i = height_o - 1;

  float Pvalue = 0.0f;
  if ((row_i >= 0) && (row_i < y_size) &&
      (col_i >= 0) && (col_i < x_size) &&
      (height_i >= 0) && (height_i < z_size)) {
    tile[tz][ty][tx] = input[height_i * y_size * x_size + row_i * x_size + col_i];
  } else {
    tile[tz][ty][tx] = 0.0f;
  }
  __syncthreads (); // wait for tile
  if (ty < TILE_WIDTH && tx < TILE_WIDTH && tz < TILE_WIDTH) {
    for(int i = 0; i < MASK_WIDTH; i++) {
      for(int j = 0; j < MASK_WIDTH; j++) {
        for(int k = 0; k < MASK_WIDTH; k++) {
          Pvalue += Mc[i][j][k] * tile[i+tz][j+ty][k+tx];
        }
      }
    }
    if (height_o < z_size && row_o < y_size && col_o < x_size) {
      output[height_o * y_size * x_size + row_o * x_size + col_o] = Pvalue;
    }
  } 
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  float *deviceInput;
  float *deviceOutput;
  //@@ Initial deviceInput and deviceOutput here.

  args = wbArg_read(argc, argv);

  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel =
      (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);


  //@@ Allocate GPU memory here
  // Recall that inputLength is 3 elements longer than the input data
  // because the first  three elements were the dimensions
  hipMalloc((void **) &deviceInput, (inputLength - 3) * sizeof(float));
  hipMalloc((void **) &deviceOutput, (inputLength - 3) * sizeof(float));


  //@@ Copy input and kernel to GPU here
  // Recall that the first three elements of hostInput are dimensions and
  // do
  // not need to be copied to the gpu
  hipMemcpy(deviceInput, &hostInput[3], (inputLength - 3) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(Mc), hostKernel, kernelLength * sizeof(float));


  //@@ Initialize grid and block dimensions here
  dim3 DimGrid(ceil(((float)x_size) / TILE_WIDTH), ceil(((float)y_size) / TILE_WIDTH), ceil(((float)z_size) / TILE_WIDTH));
  dim3 DimBlock(TILE_SIZE, TILE_SIZE, TILE_SIZE);
  //@@ Launch the GPU kernel here
  conv3d<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, z_size, y_size, x_size);
  hipDeviceSynchronize();



  //@@ Copy the device memory back to the host here
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)
  hipMemcpy(hostOutput + 3, deviceOutput, (inputLength - 3) * sizeof(float), hipMemcpyDeviceToHost);



  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);

  //@@ Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}

