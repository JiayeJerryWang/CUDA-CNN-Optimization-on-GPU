#include "hip/hip_runtime.h"
// MP5 Reduction
// Input: A num list of length n
// Output: Sum of the list = list[0] + list[1] + ... + list[n-1];

#include <wb.h>

#define BLOCK_SIZE 512 //@@ This value is not fixed and you can adjust it according to the situation

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)
  
__global__ void total(float *input, float *output, int len) {
  //@@ Load a segment of the input vector into shared memory
  //@@ Traverse the reduction tree
  //@@ Write the computed sum of the block to the output vector at the correct index
  __shared__ float partialSum[2*BLOCK_SIZE];
  unsigned int t = threadIdx.x;
  unsigned int start = 2*blockIdx.x*blockDim.x;
  partialSum[t] = input[start + t];
  partialSum[blockDim.x+t] = input[start+ blockDim.x+t];
  for (unsigned int stride = blockDim.x; stride >= 1; stride >>= 1) {
    __syncthreads();
    if (t < stride) {
      partialSum[t] += partialSum[t+stride];
    }
  }
  if (t == 0) {
    output[blockIdx.x] = partialSum[0];
  }
}

int main(int argc, char **argv) {
  int ii;
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  //@@ Initialize device input and output pointers
  float *deviceInput;
  float *deviceOutput;
  int numInputElements;  // number of elements in the input list
  int numOutputElements; // number of elements in the output list

  args = wbArg_read(argc, argv);

  //Import data and create memory on host
  hostInput =
      (float *)wbImport(wbArg_getInputFile(args, 0), &numInputElements);

  numOutputElements = numInputElements / (BLOCK_SIZE << 1);
  if (numInputElements % (BLOCK_SIZE << 1)) {
    numOutputElements++;
  }
  hostOutput = (float *)malloc(numOutputElements * sizeof(float));

  // The number of input elements in the input is numInputElements
  // The number of output elements in the input is numOutputElements

  //@@ Allocate GPU memory
  hipMalloc((void**) &deviceInput,  numInputElements * sizeof(float));
  hipMalloc((void**) &deviceOutput, numOutputElements * sizeof(float));

  //@@ Copy input memory to the GPU
  hipMemcpy(deviceInput, hostInput, numInputElements * sizeof(float), hipMemcpyHostToDevice);

  //@@ Initialize the grid and block dimensions here
  dim3 dimGrid(numOutputElements, 1, 1);
  dim3 dimBlock(BLOCK_SIZE, 1, 1);

  //@@ Launch the GPU Kernel and perform CUDA computation
  total<<<dimGrid, dimBlock>>> (deviceInput, deviceOutput, numInputElements);
  
  hipDeviceSynchronize();  
  //@@ Copy the GPU output memory back to the CPU
  hipMemcpy(hostOutput, deviceOutput, numOutputElements * sizeof(float), hipMemcpyDeviceToHost);
  
  /********************************************************************
   * Reduce output vector on the host
   * NOTE: One could also perform the reduction of the output vector
   * recursively and support any size input. 
   * For simplicity, we do not require that for this lab.
   ********************************************************************/
  for (ii = 1; ii < numOutputElements; ii++) {
    hostOutput[0] += hostOutput[ii];
  }

  //@@ Free the GPU memory
  hipFree(deviceInput);
  hipFree(deviceOutput);


  wbSolution(args, hostOutput, 1);

  free(hostInput);
  free(hostOutput);

  return 0;
}

